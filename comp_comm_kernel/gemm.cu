#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(const float* A, const float* B, float* C, int M, int N, int K) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) 
    {
        float value = 0;
        for (int i = 0; i < K; ++i) 
        {
            value += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = value;
    }
}

int main(int argc, char* argv[]) 
{
    // matrix dimensions
    int M = 512;
    int N = 512; 
    int K = 512; 

    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);

    // allocate memory
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);

    // create a random matrix
    auto rand_max = static_cast<float>(RAND_MAX);
    for (int i = 0; i < M * K; ++i)
    { 
        h_A[i] = static_cast<float>(rand()) / rand_max;
    }
    for (int i = 0; i < K * N; ++i)
    {
         h_B[i] = static_cast<float>(rand()) / rand_max;
    }

    // allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    // setup comms
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                  (M + blockSize.y - 1) / blockSize.y); 

    // launch CUDA kernel
    matrixMulKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);

    // wait for the GPU to finish
    hipDeviceSynchronize();

    // Device-to-Host (D2H) communication
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // validate
    bool correct = true;
    for (int i = 0; i < M && correct; ++i) 
    {
        for (int j = 0; j < N && correct; ++j) 
        {
            float expected = 0;
            for (int k = 0; k < K; ++k)
            {
                expected += h_A[i * K + k] * h_B[k * N + j];
            }
            if (fabs(h_C[i * N + j] - expected) > 1e-5) 
            {
                correct = false;
                std::cout << "Mismatch at (" << i << ", " << j << "): "
                          << "GPU result = " << h_C[i * N + j]
                          << ", Expected = " << expected << "\n";
            }
        }
    }

    if (correct) 
    {
        std::cout << "Matrix multiplication successful!\n";
    } 
    else
    {
        std::cout << "Matrix multiplication failed!\n";
    }

    // free all memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
