#include <iostream>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

// Optimized CUDA kernel for tiled GEMM
__global__ void tiledMatrixMulKernel(const float* A, const float* B, float* C, int M, int N, int K) 
{
    // Shared memory for tiles of A and B
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Calculate row and column indices for the current thread
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float value = 0.0f;

    // Loop over all tiles of A and B required to compute C[row][col]
    for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) 
    {
        // Load elements into shared memory (with boundary check)
        if (row < M && t * TILE_SIZE + threadIdx.x < K)
        {
            tileA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        }
        else
        {
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < K)
        {
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        }
        else
        {
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads(); // Synchronize threads to ensure all tiles are loaded

        // Compute partial product for this tile
        for (int i = 0; i < TILE_SIZE; ++i) 
        {
            value += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }

        __syncthreads(); // Synchronize before loading the next tile
    }

    // Write the result to global memory (with boundary check)
    if (row < M && col < N) 
    {
        C[row * N + col] = value;
    }
}

int main(int argc, char* argv[]) 
{
    // matrix dimensions
    int M = 512;
    int N = 512; 
    int K = 512; 

    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);

    // allocate memory
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);

    // create a random matrix
    auto rand_max = static_cast<float>(RAND_MAX);
    for (int i = 0; i < M * K; ++i)
    { 
        h_A[i] = static_cast<float>(rand()) / rand_max;
    }
    for (int i = 0; i < K * N; ++i)
    {
         h_B[i] = static_cast<float>(rand()) / rand_max;
    }

    // allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    // setup comms
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                  (M + blockSize.y - 1) / blockSize.y); 

    // launch CUDA kernel
    tiledMatrixMulKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, N, K);

    // wait for the GPU to finish
    hipDeviceSynchronize();

    // Device-to-Host (D2H) communication
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // validate
    bool correct = true;
    for (int i = 0; i < M && correct; ++i) 
    {
        for (int j = 0; j < N && correct; ++j) 
        {
            float expected = 0;
            for (int k = 0; k < K; ++k)
            {
                expected += h_A[i * K + k] * h_B[k * N + j];
            }
            if (fabs(h_C[i * N + j] - expected) > 1e-5) 
            {
                correct = false;
                std::cout << "Mismatch at (" << i << ", " << j << "): "
                          << "GPU result = " << h_C[i * N + j]
                          << ", Expected = " << expected << "\n";
            }
        }
    }

    if (correct) 
    {
        std::cout << "Matrix multiplication successful!\n";
    } 
    else
    {
        std::cout << "Matrix multiplication failed!\n";
    }

    // free all memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
