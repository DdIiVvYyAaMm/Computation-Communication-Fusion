#include <hip/hip_runtime.h> // Required for CUDA runtime API
#include <stdio.h>        // Required for printf

// CUDA kernel
__global__ void helloWorldKernel() {
    printf("Hello, World from GPU thread %d!\n", threadIdx.x);
}

int main() {
    // Launch the kernel with 1 block and 10 threads
    helloWorldKernel<<<1, 10>>>();

    // Wait for the GPU to finish before accessing results
    hipDeviceSynchronize();

    return 0;
}